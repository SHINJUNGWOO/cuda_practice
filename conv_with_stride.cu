#include "hip/hip_runtime.h"


#include "model.cuh"


__constant__  int image_size[3];
__constant__  int kernel_size[4];
__constant__  int pad[1];
__constant__  int stride[1];


__device__ void flat_conv(float* Input, float* Kernel, float* Output,/* int* image_size, int* kernel_size, int* pad,int* stride,*/int* out_w)
{
    extern __shared__ float kernel_part[];
    

    int col_idx = stride[0]*blockIdx.x - pad[0] + threadIdx.x;
    int row_idx = stride[0]*blockIdx.y - pad[0] + threadIdx.y;
    int img_flat_size = image_size[1]*image_size[2];
    int kernel_flat_size = kernel_size[2]*kernel_size[3];
    if( image_size[2]>col_idx && col_idx >=0 && image_size[1]>row_idx && row_idx >=0)
    {
        kernel_part[(threadIdx.y * kernel_size[3]+threadIdx.x)*kernel_size[1]+threadIdx.z] 
            = Input[(col_idx * image_size[2] +row_idx) + img_flat_size*threadIdx.z]
            * Kernel[threadIdx.y*kernel_size[3] + threadIdx.x + kernel_flat_size*threadIdx.z];
    }
    else
    {
        kernel_part[(threadIdx.y * kernel_size[3]+threadIdx.x)*kernel_size[1]+threadIdx.z] = 0;
    }
    //__syncthreads;

    atomicAdd(&(Output[blockIdx.x * out_w[0] +blockIdx.y]), kernel_part[(threadIdx.y * kernel_size[3]+threadIdx.x)*kernel_size[1]+threadIdx.z]);
}

__global__ void conv(float* Input, float* Kernel, float* Output/*, int* image_size, int* kernel_size, int* pad,int* stride*/)
{   
    int out_w = (image_size[2]+2*pad[0] - kernel_size[3])/stride[0] + 1;
    int out_h = (image_size[1]+2*pad[0] - kernel_size[2])/stride[0] + 1;
    int flat_kernel_size = kernel_size[3]*kernel_size[2]*kernel_size[1];
    int flat_img_size = out_w*out_h;
    flat_conv(Input, Kernel + flat_kernel_size*blockIdx.z , Output + flat_img_size*blockIdx.z, /* image_size, kernel_size, pad, stride,*/ &out_w);

}

__global__ void relu(float*Input)
{
    int col_idx = blockIdx.x* blockDim.x + threadIdx.x;
    int row_idx = blockIdx.y* blockDim.y + threadIdx.y;
    if ( (Input + blockIdx.z * blockDim.x*blockDim.y)[row_idx*gridDim.x*blockDim.x + col_idx]<0)
    {
        (Input + blockIdx.z * blockDim.x*blockDim.y)[row_idx*gridDim.x*blockDim.x + col_idx] = 0;
    }
}

__host__ float* convolution_relu(float* Input, float* Kernel, int* h_image_size, int* h_kernel_size, int h_pad,int h_stride)
{
    int out_w = (h_image_size[2]+2*h_pad - h_kernel_size[3])/h_stride + 1;
    int out_h = (h_image_size[1]+2*h_pad - h_kernel_size[2])/h_stride + 1;
    int flat_kernel_size = h_kernel_size[3]* h_kernel_size[2]* h_kernel_size[1]*sizeof(float);
    float* Output;
    
    hipMemcpyToSymbol(HIP_SYMBOL(image_size),h_image_size,sizeof(int)*3);
    hipMemcpyToSymbol(HIP_SYMBOL(kernel_size),h_kernel_size,sizeof(int)*4);
    hipMemcpyToSymbol(HIP_SYMBOL(pad),&h_pad,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(stride),&h_stride,sizeof(int));
    hipMalloc((void***)&Output,out_w*out_h*h_kernel_size[0]*sizeof(float));
    
    dim3 threads_c(h_kernel_size[3], h_kernel_size[2], h_kernel_size[1]);
	dim3 grid_c(out_w,out_h,h_kernel_size[0]);

    conv <<< grid_c,threads_c,flat_kernel_size>>>(Input,Kernel,Output);

    int relu_t_wsize = h_image_size[2] > 32 ? 32: out_w;
    int relu_t_hsize = h_image_size[1] > 32 ? 32: out_h;
    dim3 threads_r(relu_t_wsize,relu_t_hsize);
	dim3 grid_r(out_w/relu_t_wsize,out_h/relu_t_hsize,h_kernel_size[0]);
    relu <<<grid_r,threads_r >>>(Output);
    
    hipFree(Input);
    //hipFree(Kernel);
    return Output;
}

__global__ void g_maxpooling(float* Input,float* Output)
{
    
    int row_idx = blockIdx.y*blockDim.y +threadIdx.y;
    int col_idx = blockIdx.x*blockDim.x +threadIdx.x;
    int col_len = gridDim.x*blockDim.x;
    int row_len = gridDim.y*blockDim.x;
    int flat_img_size = col_len*row_len;
    extern __shared__ float kernel_part[];
    //size of double of kernel

    int shm_idx = blockDim.x*threadIdx.y + threadIdx.x +blockDim.x*blockDim.y*threadIdx.z;

    kernel_part[shm_idx] = Input[row_idx*col_len + col_idx + threadIdx.z*flat_img_size];
    __syncthreads();
    for (int size = blockDim.x/2; size>0; size=size/2) { 
        if (threadIdx.x < size)
        {
            kernel_part[shm_idx] =  kernel_part[shm_idx] >
                                    kernel_part[shm_idx + size] ?
                                    kernel_part[shm_idx] :
                                    kernel_part[shm_idx + size] ;
           

        } 
        __syncthreads();
    }
    for (int size = blockDim.y/2; size>0; size=size/2) { 
        if (threadIdx.y < size)
        {
            kernel_part[shm_idx] =  kernel_part[shm_idx] >
                                    kernel_part[shm_idx + size*blockDim.x] ?
                                    kernel_part[shm_idx] :
                                    kernel_part[shm_idx + size*blockDim.x] ;
            
        }
        __syncthreads();

    }
    if(threadIdx.x == 0 && threadIdx.y == 0)
    {
        Output[blockIdx.y*gridDim.x+blockIdx.x + threadIdx.z*gridDim.x*gridDim.y] = kernel_part[blockDim.x*blockDim.y*threadIdx.z];
    }
}
__host__ float* maxpooling(float* Input, int* h_image_size, int h_kernel_size)
{
    float* Output;
    int col_len = h_image_size[2]/h_kernel_size;
    int row_len = h_image_size[1]/h_kernel_size;
    hipMalloc((void***)&Output,row_len*col_len*h_image_size[0]*sizeof(float));
    dim3 threads(h_kernel_size,h_kernel_size,h_image_size[0]);
	dim3 grid(col_len,row_len);
    int shm_len = h_image_size[0]*h_kernel_size*h_kernel_size*sizeof(float);

    g_maxpooling<<<grid,threads,shm_len>>>(Input,Output);

    hipFree(Input);
    return Output;

}


__global__ void fc(float* Input,float* Kernel, float* Output)
{
    extern __shared__ float kernel_part[];
    kernel_part[threadIdx.x] = Input[threadIdx.x] * Kernel[blockDim.x*blockIdx.x+threadIdx.x];

    atomicAdd(&Output[blockIdx.x],kernel_part[threadIdx.x]);
}

__host__ float* fully_connected(float* Input,float* Kernel,int* kernel_size)
{   // Weight size = (Out channel, In channel)
    
    float* Output;
    hipMalloc((void**)&Output,kernel_size[0]*sizeof(float));
    int shm_len = kernel_size[1]*sizeof(float);
    dim3 thread(kernel_size[1]);
    dim3 grid(kernel_size[0]);
    fc<<<grid,thread,shm_len>>>(Input,Kernel,Output);
    return Output;
}




/*
__host__ int main()
{
    float* h_img; float *h_kernel; float* h_out;
    int h_img_len = sizeof(float)*10;
    int h_kernel_len = sizeof(float)*10*10;
    int h_out_len = sizeof(float)*10;

    h_img = (float*)malloc(h_img_len);
    h_kernel = (float*)malloc(h_kernel_len);
    h_out = (float*)malloc(h_out_len);
    OneInit(h_kernel,h_kernel_len/sizeof(float));
    OneInit(h_img,h_img_len/sizeof(float));
    h_kernel[0] = 10;
    float *cimg;
    float *ckernel;
    hipMalloc((void**)&cimg,h_img_len);
    hipMalloc((void**)&ckernel,h_kernel_len);
    hipMemcpy(cimg,h_img,h_img_len,hipMemcpyHostToDevice);
    hipMemcpy(ckernel,h_kernel,h_kernel_len,hipMemcpyHostToDevice);
    int w_s[2] = {10,10};
    float *cout;
    cout = fully_connected(cimg,ckernel,w_s);
    hipMemcpy(h_out,cout,h_out_len,hipMemcpyDeviceToHost);
    for(int i =0; i <10; i++)
    {
        printf("%.0f ",h_out[i]);
    }


}
*/