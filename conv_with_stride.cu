#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define IMW 64   
#define IMH 64     
#define IMC 3 
 
#define WO 3
#define WI 3
#define WH 3
#define WW 3

#define PAD 1
#define STIRDE 2

#define OIMW ((IMW+2*PAD-WW)/STIRDE+1) 
#define OIMH ((IMH+2*PAD-WH)/STIRDE+1) 

__constant__  int image_size[3];
__constant__  int kernel_size[4];
__constant__  int pad[1];
__constant__  int stride[1];


__device__ void flat_conv(float* Input, float* Kernel, float* Output,/* int* image_size, int* kernel_size, int* pad,int* stride,*/int* out_w)
{
    extern __shared__ float kernel_part[];
    

    int col_idx = stride[0]*blockIdx.x - pad[0] + threadIdx.x;
    int row_idx = stride[0]*blockIdx.y - pad[0] + threadIdx.y;
    int img_flat_size = image_size[1]*image_size[2];
    int kernel_flat_size = kernel_size[2]*kernel_size[3];
    if( image_size[2]>col_idx && col_idx >=0 && image_size[1]>row_idx && row_idx >=0)
    {
        kernel_part[(threadIdx.y * kernel_size[3]+threadIdx.x)*kernel_size[1]+threadIdx.z] 
            = Input[(col_idx * image_size[2] +row_idx) + img_flat_size*threadIdx.z]
            * Kernel[threadIdx.y*kernel_size[3] + threadIdx.x + kernel_flat_size*threadIdx.z];
    }
    else
    {
        kernel_part[(threadIdx.y * kernel_size[3]+threadIdx.x)*kernel_size[1]+threadIdx.z] = 0;
    }
    //__syncthreads;

    atomicAdd(&(Output[blockIdx.x * out_w[0] +blockIdx.y]), kernel_part[(threadIdx.y * kernel_size[3]+threadIdx.x)*kernel_size[1]+threadIdx.z]);
}

__global__ void conv(float* Input, float* Kernel, float* Output/*, int* image_size, int* kernel_size, int* pad,int* stride*/)
{   
    int out_w = (image_size[2]+2*pad[0] - kernel_size[3])/stride[0] + 1;
    int out_h = (image_size[1]+2*pad[0] - kernel_size[2])/stride[0] + 1;
    int flat_kernel_size = kernel_size[3]*kernel_size[2]*kernel_size[1];
    int flat_img_size = out_w*out_h;
    flat_conv(Input, Kernel + flat_kernel_size*blockIdx.z , Output + flat_img_size*blockIdx.z, /* image_size, kernel_size, pad, stride,*/ &out_w);

}

__global__ void relu(float*Input)
{
    int col_idx = blockIdx.x* blockDim.x + threadIdx.x;
    int row_idx = blockIdx.y* blockDim.y + threadIdx.y;
    if ( (Input + blockIdx.z * blockDim.x*blockDim.y)[row_idx*gridDim.x*blockDim.x + col_idx]<0)
    {
        (Input + blockIdx.z * blockDim.x*blockDim.y)[row_idx*gridDim.x*blockDim.x + col_idx] = 0;
    }
}

__host__ float* convolution_relu(float* Input, float* Kernel, int* h_image_size, int* h_kernel_size, int h_pad,int h_stride)
{
    int out_w = (h_image_size[2]+2*h_pad - h_kernel_size[3])/h_stride + 1;
    int out_h = (h_image_size[1]+2*h_pad - h_kernel_size[2])/h_stride + 1;
    int flat_kernel_size = h_kernel_size[3]* h_kernel_size[2]* h_kernel_size[1]*sizeof(float);
    float* Output;
    
    hipMemcpyToSymbol(HIP_SYMBOL(image_size),h_image_size,sizeof(int)*3);
    hipMemcpyToSymbol(HIP_SYMBOL(kernel_size),h_kernel_size,sizeof(int)*4);
    hipMemcpyToSymbol(HIP_SYMBOL(pad),&h_pad,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(stride),&h_stride,sizeof(int));
    hipMalloc((void***)&Output,out_w*out_h*h_kernel_size[0]*sizeof(float));
    
    dim3 threads_c(h_kernel_size[3], h_kernel_size[2], h_kernel_size[1]);
	dim3 grid_c(out_w,out_h,h_kernel_size[0]);

    conv <<< grid_c,threads_c,flat_kernel_size>>>(Input,Kernel,Output);

    dim3 threads_r(32,32);
	dim3 grid_r(out_w/32,out_h/32,h_kernel_size[0]);
    relu <<<grid_r,threads_r >>>(Output);
    
    hipFree(Input);
    //cudaFree(Kernel);
    return Output;
}

__global__ void g_maxpooling(float* Input,float* Output)
{
    
    int row_idx = blockIdx.y*blockDim.y +threadIdx.y;
    int col_idx = blockIdx.x*blockDim.x +threadIdx.x;
    int col_len = gridDim.x*blockDim.x;
    int row_len = gridDim.y*blockDim.x;
    int flat_img_size = col_len*row_len;
    extern __shared__ float kernel_part[];
    //size of double of kernel

    int shm_idx = blockDim.x*threadIdx.y + threadIdx.x +blockDim.x*blockDim.y*threadIdx.z;

    kernel_part[shm_idx] = Input[row_idx*col_len + col_idx + threadIdx.z*flat_img_size];
    __syncthreads();
    for (int size = blockDim.x/2; size>0; size=size/2) { 
        if (threadIdx.x < size)
        {
            kernel_part[shm_idx] =  kernel_part[shm_idx] >
                                    kernel_part[shm_idx + size] ?
                                    kernel_part[shm_idx] :
                                    kernel_part[shm_idx + size] ;
           

        } 
        __syncthreads();
    }
    for (int size = blockDim.y/2; size>0; size=size/2) { 
        if (threadIdx.y < size)
        {
            kernel_part[shm_idx] =  kernel_part[shm_idx] >
                                    kernel_part[shm_idx + size*blockDim.x] ?
                                    kernel_part[shm_idx] :
                                    kernel_part[shm_idx + size*blockDim.x] ;
            
        }
        __syncthreads();

    }
    if(threadIdx.x == 0 && threadIdx.y == 0)
    {
        Output[blockIdx.y*gridDim.x+blockIdx.x + threadIdx.z*gridDim.x*gridDim.y] = kernel_part[blockDim.x*blockDim.y*threadIdx.z];
    }
}
__host__ float* maxpooling(float* Input, int* h_image_size, int h_kernel_size)
{
    float* Output;
    int col_len = h_image_size[2]/h_kernel_size;
    int row_len = h_image_size[1]/h_kernel_size;
    hipMalloc((void***)&Output,row_len*col_len*h_image_size[0]*sizeof(float));
    dim3 threads(h_kernel_size,h_kernel_size,h_image_size[0]);
	dim3 grid(col_len,row_len);
    int shm_len = h_image_size[0]*h_kernel_size*h_kernel_size*sizeof(float);

    g_maxpooling<<<grid,threads,shm_len>>>(Input,Output);

    hipFree(Input);
    return Output;

}


//// HOST /////
void randomInit(float* data, int size)
{
    
	for (int i = 0; i < size; ++i)
		data[i] = (rand() / (float)RAND_MAX) -0.5;
}
void OneInit(float* data, int size)
{
    
	for (int i = 0; i < size; ++i)
		data[i] = 1;
}
__host__ int main(void)
{

    int h_kernel_size[4] ={WO,WI,WH,WW}; //O I H W;
    int h_image_size[3] = {IMC,IMH,IMW}; //  O H W;

    float* h_img; float *h_kernel; float* h_out;

    int h_img_len = sizeof(float)*IMC*IMH*IMW;
    int h_kernel_len = sizeof(float)*WO*WI*WH*WW;
    int h_out_len = sizeof(float)*WO*OIMH*OIMW;

    h_img = (float*)malloc(h_img_len);
    h_kernel = (float*)malloc(h_kernel_len);
    h_out = (float*)malloc(h_out_len);
    
    //h_img[400] = 1; h_img[400+IMH*IMW] =1; h_img[400+2*IMH*IMW] =1;
    randomInit(h_img,h_img_len/sizeof(float));
    OneInit(h_kernel,h_kernel_len/sizeof(float));
    int h_pad = PAD;
    int h_stride = STIRDE;



    float *cimg;
    
    float *ckernel;
    hipMalloc((void***)&cimg,h_img_len);
    hipMalloc((void***)&ckernel,h_kernel_len);

    hipMemcpy(cimg,h_img,h_img_len,hipMemcpyHostToDevice);
    hipMemcpy(ckernel,h_kernel,h_kernel_len,hipMemcpyHostToDevice);

    clock_t start = clock();
    float *coimg_1;
    coimg_1 = convolution_relu(cimg,ckernel,h_image_size,h_kernel_size,1,1);
    float *coimg_2;
    coimg_2 = maxpooling(coimg_1,h_image_size,2);
    
    h_image_size[0] = 3;h_image_size[1] = 32;h_image_size[2] = 32;
    
    float *coimg_3;
    coimg_3 = convolution_relu(coimg_2,ckernel,h_image_size,h_kernel_size,1,1);
    clock_t end = clock();
    hipMemcpy(h_out,coimg_3,h_out_len,hipMemcpyDeviceToHost);

    int cnt = 0;
    for(int i = 0;i < WO; i++)
    {
        for(int j =0; j < OIMH;j ++)
        {
            for(int k =0; k < OIMW;k ++)
            {
                //printf("%.0f ",h_c[cnt]);
                printf("%.1f ",h_out[cnt]);
                cnt +=1;
            }
            printf("\n");   
        }
        printf("\n");
    }

    //cudaFree(cimg);
    hipFree(ckernel);
    hipFree(coimg_2);
    //cudaFree(cimg_size);
    //cudaFree(ckernel_size);
    //cudaFree(cpad);
    //cudaFree(cstride);
    
    
    printf("%f",(float)(end - start)/CLOCKS_PER_SEC);
}