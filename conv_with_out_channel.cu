#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define BLOCK_SIZE 32
#define WA 64   
#define HA 64     
#define HC 3     
#define WC 3
#define PAD 1
#define WB (WA+2*PAD - WC + 1)
#define HB (HA+2*PAD - HC + 1)
#define CHANNEL_SIZE 3

__device__ void flat_conv(float* Input, float* Kernel, float* Output,int* image_size, int* kernel_size, int* pad,int* out_w)
{
    //__shared__ float kernel_part[kernel_size[2]][kernel_size[3]][kernel_size[1]];
    //__shared__ float kernel_part[3][3][3];
    extern __shared__ float kernel_part[];
    

    int col_idx = blockIdx.x - pad[0] + threadIdx.x;
    int row_idx = blockIdx.y - pad[0] + threadIdx.y;
    int img_flat_size = image_size[1]*image_size[2];
    int kernel_flat_size = kernel_size[2]*kernel_size[3];
    if( image_size[2]>col_idx && col_idx >=0 && image_size[1]>row_idx && row_idx >=0)
    {
        kernel_part[(threadIdx.y * kernel_size[3]+threadIdx.x)*kernel_size[1]+threadIdx.z] 
            = Input[(col_idx * image_size[2] +row_idx) + img_flat_size*threadIdx.z]
            * Kernel[threadIdx.y*kernel_size[3] + threadIdx.x + kernel_flat_size*threadIdx.z];
    }
    else
    {
        kernel_part[(threadIdx.y * kernel_size[3]+threadIdx.x)*kernel_size[1]+threadIdx.z] = 0;
    }
    //__syncthreads;

    atomicAdd(&(Output[blockIdx.x * out_w[0] +blockIdx.y]), kernel_part[(threadIdx.y * kernel_size[3]+threadIdx.x)*kernel_size[1]+threadIdx.z]);
}


__global__ void conv(float* Input, float* Kernel, float* Output,int* image_size, int* kernel_size,int* pad)
{   
    int out_w = image_size[2]+2*pad[0] - kernel_size[3] + 1;
    int out_h = image_size[1]+2*pad[0] - kernel_size[2] + 1;
    int flat_kernel_size = kernel_size[3]*kernel_size[2]*kernel_size[1];
    int flat_img_size = out_w*out_h;
    flat_conv(Input, Kernel + flat_kernel_size*blockIdx.z , Output + flat_img_size*blockIdx.z, image_size, kernel_size, pad,&out_w);
}


__host__ int main(void)
{

    float h_a[3][64][64] ={0.0};
    h_a[0][0][0] = 2.1;
    h_a[1][0][0] = 2.1;
    h_a[2][0][0] = 2.1;
    float h_b[2][3][3][3] ={1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,
                            1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,
                            1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,
                            2.0,2.0,2.0,2.0,2.0,2.0,2.0,2.0,2.0,
                            2.0,2.0,2.0,2.0,2.0,2.0,2.0,2.0,2.0,
                            2.0,2.0,2.0,2.0,2.0,2.0,2.0,2.0,2.0};

    float h_c[2][64][64] ={0.0};
    int kernel_size[4] ={2,3,3,3}; //O I H W;
    int image_size[3] = {3,64,64}; //  O H W;
    int pad = 1;

    float *cimg;
    float *coimg;
    float *ckernel;
    int * cimg_size;
    int * ckernel_size;
    int * cpad;
    hipMalloc((void***)&cimg,sizeof(h_a));
    hipMalloc((void***)&ckernel,sizeof(h_b));
    hipMalloc((void***)&coimg,sizeof(h_c));
    hipMalloc(&cimg_size,sizeof(image_size));
    hipMalloc(&ckernel_size,sizeof(kernel_size));
    hipMalloc(&cpad,sizeof(int));

    hipMemcpy(cimg,h_a,sizeof(h_a),hipMemcpyHostToDevice);
    hipMemcpy(ckernel,h_b,sizeof(h_b),hipMemcpyHostToDevice);
    hipMemcpy(cimg_size,image_size,sizeof(image_size),hipMemcpyHostToDevice);
    hipMemcpy(ckernel_size,kernel_size,sizeof(kernel_size),hipMemcpyHostToDevice);
    hipMemcpy(cpad,&pad,sizeof(int),hipMemcpyHostToDevice);

    dim3 threads(kernel_size[3], kernel_size[2], kernel_size[1]);
	dim3 grid(image_size[2],image_size[1],kernel_size[0]);
    clock_t start = clock(); 

    int flat_kernel_size = kernel_size[3]* kernel_size[2]* kernel_size[1];
    conv <<< grid,threads,flat_kernel_size>>>(cimg,ckernel,coimg,cimg_size,ckernel_size,cpad);
    //Convolution <<< grid,threads>>>(cimg,ckernel,coimg,cimg_size,ckernel_size);

    clock_t end = clock();
    hipMemcpy(h_c,coimg,sizeof(h_c),hipMemcpyDeviceToHost);

    for(int j =0; j < WB;j ++)
    {
        for(int k =0; k < WB;k ++)
        {
            printf("%.1f ",h_c[1][k][j]);
        }
        printf("\n");   
    }
    printf("\n");


    printf("%f",(float)(end - start)/CLOCKS_PER_SEC);
}